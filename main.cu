/***************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 *cr   This version maintained by: Nasser Anssari (anssari1@illinois.edu)
 ***************************************************************************/

 #include <stdio.h>
 #include <stdlib.h>
 #include <hip/hip_runtime.h>
 
 #include "file.h"
 #include "kernel.h"
 
 void readVectorBinary(const char* filename, float** vector, unsigned* size) {
     FILE* file = fopen(filename, "rb");
     if (file == NULL) {
         fprintf(stderr, "Error opening file %s\n", filename);
         exit(EXIT_FAILURE);
     }
 
     // Read the size of the vector
     fread(size, sizeof(unsigned), 1, file);
 
     // Allocate memory for the vector
     *vector = (float*)malloc(*size * sizeof(float));
     if (*vector == NULL) {
         fprintf(stderr, "Unable to allocate memory for vector %s\n", filename);
         exit(EXIT_FAILURE);
     }
 
     // Read the vector data
     fread(*vector, sizeof(float), *size, file);
 
     fclose(file);
 }
 
 void writeVectorBinary(const char* filename, const float* vector, unsigned size) {
     FILE* file = fopen(filename, "wb");
     if (file == NULL) {
         fprintf(stderr, "Error opening file %s\n", filename);
         exit(EXIT_FAILURE);
     }
 
     // Write the size of the vector
     fwrite(&size, sizeof(unsigned), 1, file);
 
     // Write the vector data
     fwrite(vector, sizeof(float), size, file);
 
     fclose(file);
 }
 
 int main(int argc, char *argv[]) {
     float *A_h, *B_h, *C_h;
     float *A_d, *B_d, *C_d;
     unsigned vec_size;
     hipError_t cuda_ret;
     dim3 dim_grid, dim_block;
 
     if (argc != 3) {
         fprintf(stderr, "Usage: %s <input1.dat> <input2.dat>\n", argv[0]);
         return EXIT_FAILURE;
     }
 
     /* Initialize input vectors */
     readVectorBinary(argv[1], &A_h, &vec_size);
     readVectorBinary(argv[2], &B_h, &vec_size);
 
     printf("Vector size: %u\n", vec_size);
 
     /* Allocate host memory */
     C_h = (float *)malloc(vec_size * sizeof(float));
     if(C_h == NULL) {
         fprintf(stderr, "Unable to allocate host memory for C_h\n");
         return EXIT_FAILURE;
     }
 
     /********************************************************************
     Allocate device memory for the input/output vectors
     ********************************************************************/
     cuda_ret = hipMalloc((void**)&A_d, vec_size * sizeof(float));
     if (cuda_ret != hipSuccess) {
         fprintf(stderr, "Unable to allocate device memory for A\n");
         return EXIT_FAILURE;
     }
     
     cuda_ret = hipMalloc((void**)&B_d, vec_size * sizeof(float));
     if (cuda_ret != hipSuccess) {
         fprintf(stderr, "Unable to allocate device memory for B\n");
         return EXIT_FAILURE;
     }
     
     cuda_ret = hipMalloc((void**)&C_d, vec_size * sizeof(float));
     if (cuda_ret != hipSuccess) {
         fprintf(stderr, "Unable to allocate device memory for C\n");
         return EXIT_FAILURE;
     }
 
     /********************************************************************
     Copy the input vectors from the host memory to the device memory
     ********************************************************************/
     cuda_ret = hipMemcpy(A_d, A_h, vec_size * sizeof(float), hipMemcpyHostToDevice);
     if (cuda_ret != hipSuccess) {
         fprintf(stderr, "Unable to copy memory to device for A\n");
         return EXIT_FAILURE;
     }
     
     cuda_ret = hipMemcpy(B_d, B_h, vec_size * sizeof(float), hipMemcpyHostToDevice);
     if (cuda_ret != hipSuccess) {
         fprintf(stderr, "Unable to copy memory to device for B\n");
         return EXIT_FAILURE;
     }
 
     cuda_ret = hipMemset(C_d, 0, vec_size * sizeof(float));
     if(cuda_ret != hipSuccess) {
         fprintf(stderr, "Unable to set device memory\n");
         return EXIT_FAILURE;
     }
 
     /********************************************************************
     Initialize thread block and kernel grid dimensions
     ********************************************************************/
     int threads_per_block = 512;
     int num_blocks = (vec_size + threads_per_block - 1) / threads_per_block;
 
     dim_block = dim3(threads_per_block, 1, 1);
     dim_grid = dim3(num_blocks, 1, 1);
 
     /********************************************************************
     Invoke CUDA kernel
     ********************************************************************/
     vecAdd<<<dim_grid, dim_block>>>(C_d, A_d, B_d, vec_size);
     cuda_ret = hipGetLastError();
     if(cuda_ret != hipSuccess) {
         fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cuda_ret));
         return EXIT_FAILURE;
     }
 
     cuda_ret = hipDeviceSynchronize();
     if(cuda_ret != hipSuccess) {
         fprintf(stderr, "Kernel execution failed: %s\n", hipGetErrorString(cuda_ret));
         return EXIT_FAILURE;
     }
 
     printf("Kernel executed successfully\n");
 
     /********************************************************************
     Copy the result back to the host
     ********************************************************************/
     cuda_ret = hipMemcpy(C_h, C_d, vec_size * sizeof(float), hipMemcpyDeviceToHost);
     if (cuda_ret != hipSuccess) {
         fprintf(stderr, "Unable to copy memory to host for C\n");
         return EXIT_FAILURE;
     }
 
     printf("Memory copied back to host successfully\n");
 
     /********************************************************************
     Print the first 10 values of C_h
     ********************************************************************/
     for (unsigned i = 0; i < 10 && i < vec_size; ++i) {
         printf("C_h[%u] = %f\n", i, C_h[i]);
     }
 
     /********************************************************************
     Write the result vector to a binary file for verification
     ********************************************************************/
     writeVectorBinary("output.dat", C_h, vec_size);
 
     /********************************************************************
     Free device memory allocations
     ********************************************************************/
     hipFree(A_d);
     hipFree(B_d);
     hipFree(C_d);
 
     free(A_h);
     free(B_h);
     free(C_h);
 
     return 0;
 }
 